#include "app.cuh"

#include "common.cuh"
#include "glad/gl.h"
#include "imgui.h"

#include <GLFW/glfw3.h>
#include <stdio.h>
#include <unistd.h>

static const char* vertex_shader_text = R"glsl(
#version 330 core
#extension GL_ARB_explicit_uniform_location : enable
layout(location = 0) uniform mat4 transform;

layout(location = 0) in vec2 pos;
layout(location = 1) in vec2 uv;

out vec2 uvOut;

void main()
{
    gl_Position = transform * vec4(pos, 0., 1.0);
    uvOut = uv;
}
)glsl";

static const char* fragment_shader_text = R"glsl(
#version 330 core

in vec2 uvOut;

uniform sampler2D tex;

void main()
{
    vec4 c = texture(tex, uvOut.xy);
    gl_FragColor = vec4(c.xyz, 1.0);
}
)glsl";

bool App::init() {
  glfwSetErrorCallback([](int error, const char* desc) {
    printf("ERROR: %s (%d)\n", desc, error);
  });

  if (!glfwInit()) {
    printf("ERROR: Failed to init GLFW");
    return false;
  }

  glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 2);
  glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 0);

  window = glfwCreateWindow(640, 480, "Cellular CUDA", nullptr, nullptr);
  if (!window) {
    printf("ERROR: Failed to create GLFW window");
    glfwTerminate();
    return false;
  }

  glfwSetWindowUserPointer(window, this);

  glfwMakeContextCurrent(window);
  const int gl_ver = gladLoadGL(glfwGetProcAddress);
  printf("INFO: GL version: %d.%d\n", GLAD_VERSION_MAJOR(gl_ver),
         GLAD_VERSION_MINOR(gl_ver));
  glfwSwapInterval(1);

  IMGUI_CHECKVERSION();
  ImGui::CreateContext();
  ImGuiIO& io = ImGui::GetIO();
  io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard; // Enable Keyboard Controls
  io.ConfigFlags |= ImGuiConfigFlags_NavEnableGamepad;  // Enable Gamepad Controls

  ImGui::StyleColorsDark();

  glfwSetMouseButtonCallback(window, [](GLFWwindow* window, int button, int action, int mods) {
    auto* self = (App*)glfwGetWindowUserPointer(window);
    self->on_mouse_button(button, action, mods);
  });

  glfwSetScrollCallback(window, [](GLFWwindow* window, double xoffset, double yoffset) {
    auto* self = (App*)glfwGetWindowUserPointer(window);
    self->on_scroll(xoffset, yoffset);
  });

  glfwSetCursorPosCallback(window, [](GLFWwindow* window, double x, double y) {
    auto* self = (App*)glfwGetWindowUserPointer(window);
    self->on_cursor(x, y);
  });

  ImGui_ImplGlfw_InitForOpenGL(window, true);
  if (!ImGui_ImplOpenGL3_Init("#version 130")) {
    printf("ERROR: Failed to ImGui_ImplOpenGL3_Init\n");
    return false;
  }

  glfwSetKeyCallback(window, [](GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
      glfwSetWindowShouldClose(window, GLFW_TRUE);
    }
  });

  static const struct Vertex {
    float x, y;
    float u, v;
  } vertices[6] = {
      {-1.f, -1.f, 0.f, 0.f},
      {1.f, -1.f, 1.f, 0.f},
      {1.f, 1.f, 1.f, 1.f},
      {-1.f, -1.f, 0.f, 0.f},
      {1.f, 1.f, 1.f, 1.f},
      {-1.f, 1.f, 0.f, 1.f},
  };

  glGenBuffers(1, &buf_vert);
  glBindBuffer(GL_ARRAY_BUFFER, buf_vert);
  glBufferData(GL_ARRAY_BUFFER, sizeof(vertices), vertices, GL_STATIC_DRAW);

  {
    shader_vert = glCreateShader(GL_VERTEX_SHADER);
    glShaderSource(shader_vert, 1, &vertex_shader_text, NULL);
    glCompileShader(shader_vert);

    int status;
    glGetShaderiv(shader_vert, GL_COMPILE_STATUS, &status);
    if (!status) {
      char err[512];
      glGetShaderInfoLog(shader_vert, 512, nullptr, err);
      printf("ERROR: Vertex shader compilation error:\n%s\n", err);
      return false;
    }
  }

  {
    shader_frag = glCreateShader(GL_FRAGMENT_SHADER);
    glShaderSource(shader_frag, 1, &fragment_shader_text, NULL);
    glCompileShader(shader_frag);

    int status;
    glGetShaderiv(shader_frag, GL_COMPILE_STATUS, &status);
    if (!status) {
      char err[512];
      glGetShaderInfoLog(shader_frag, 512, nullptr, err);
      printf("ERROR: Fragment shader compilation error:\n%s\n", err);
      return false;
    }
  }

  shader_prog = glCreateProgram();
  glAttachShader(shader_prog, shader_vert);
  glAttachShader(shader_prog, shader_frag);
  glLinkProgram(shader_prog);

  glEnableVertexAttribArray(0);
  glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(vertices[0]), (const void*)offsetof(Vertex, x));
  glEnableVertexAttribArray(1);
  glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, sizeof(vertices[0]), (const void*)offsetof(Vertex, u));

  CUDA_CALL(hipMalloc(&cuda_tex, tex_w * tex_h * 4));

  blocks = dim3{tex_w / threads.x, tex_h / threads.y};

  glGenTextures(1, &texture);
  glBindTexture(GL_TEXTURE_2D, texture);

  glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, tex_w, tex_h, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
  glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

  CUDA_CALL(hipGraphicsGLRegisterImage(&tex_res, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore));

  CUDA_CALL(hipGraphicsMapResources(1, &tex_res, 0));

  CUDA_CALL(hipGraphicsSubResourceGetMappedArray(&cuda_arr, tex_res, 0, 0));

  cellular.init(cuda_tex, tex_w, tex_h);

  return true;
}

bool App::term() {
  cellular.term();

  CUDA_CALL(hipGraphicsUnmapResources(1, &tex_res, 0));

  // Cleanup
  ImGui_ImplOpenGL3_Shutdown();
  ImGui_ImplGlfw_Shutdown();
  ImGui::DestroyContext();

  glfwDestroyWindow(window);
  glfwTerminate();

  return true;
}

bool App::loop() {
  ImGuiIO& io = ImGui::GetIO();

  while (!glfwWindowShouldClose(window)) {
    const double beg_time = glfwGetTime();

    glfwPollEvents();

    // Start the Dear ImGui frame
    ImGui_ImplOpenGL3_NewFrame();
    ImGui_ImplGlfw_NewFrame();
    ImGui::NewFrame();

    if (is_draw) {
      const float tx = cursor_x - as<float>(win_w) / 2;
      const float ty = cursor_y - as<float>(win_h) / 2;

      const int x = (tx - offset_x * scx) / scale;
      const int y = (ty + offset_y * scy) / scale;
      if (!cellular.draw(x + tex_w / 2, tex_h / 2 - y)) {
        return false;
      }
    }

    if (!cellular.update()) {
      return false;
    }

    CUDA_CALL(hipMemcpyToArray(cuda_arr, 0, 0, cuda_tex, tex_w * tex_h * 4, hipMemcpyDeviceToDevice));
    ImGui::Render();

    int width, height;
    glfwGetFramebufferSize(window, &width, &height);

    glViewport(0, 0, width, height);
    glClearColor(.1f, .1f, .1f, 1.f);
    glClear(GL_COLOR_BUFFER_BIT);

    // Calculate transform

    glfwGetWindowSize(window, &win_w, &win_h);

    const float ratio_w = as<float>(tex_w) / as<float>(win_w);
    const float ratio_h = as<float>(tex_h) / as<float>(win_h);

    scx = ratio_w * scale;
    scy = ratio_h * scale;

    const float dgx = (drag_x - cursor_x) / scx;
    const float dgy = (drag_y - cursor_y) / scy;

    const float psx = (offset_x - dgx) / win_w * 2 * scx;
    const float psy = (offset_y + dgy) / win_h * 2 * scy;

    float transform[4][4] = {
        {scx, 0.f, 0.f, 0.f},
        {0.f, scy, 0.f, 0.f},
        {0.f, 0.f, 1.f, 0.f},
        {psx, psy, 0.f, 1.f}};

    glUniformMatrix4fv(0, 1, false, (float*)transform);

    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, texture);

    glUseProgram(shader_prog);
    glDrawArrays(GL_TRIANGLES, 0, 6);

    ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

    glfwSwapBuffers(window);

    const double end_time = glfwGetTime();
    const double dt = end_time - beg_time;

    usleep(dt * 1e6);
  }

  return true;
}

void App::on_mouse_button(int button, int action, int mods) {
  if (!ImGui::GetIO().WantCaptureMouse && button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_PRESS) {
    is_drag = true;

    drag_x = cursor_x;
    drag_y = cursor_y;
  }

  if (button == GLFW_MOUSE_BUTTON_RIGHT && action == GLFW_RELEASE) {
    is_drag = false;

    offset_x -= (drag_x - cursor_x) / scx;
    offset_y += (drag_y - cursor_y) / scy;

    drag_x = cursor_x;
    drag_y = cursor_y;
  }

  if (!ImGui::GetIO().WantCaptureMouse && button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_PRESS) {
    is_draw = true;
  }

  if (button == GLFW_MOUSE_BUTTON_LEFT && action == GLFW_RELEASE) {
    is_draw = false;
  }
}

void App::on_scroll(float x, float y) {
  if (ImGui::GetIO().WantCaptureMouse) {
    return;
  }

  scale += y * 0.05f;
}

void App::on_cursor(float x, float y) {
  cursor_x = x;
  cursor_y = y;

  if (!is_drag) {
    drag_x = cursor_x;
    drag_y = cursor_y;
  }
}